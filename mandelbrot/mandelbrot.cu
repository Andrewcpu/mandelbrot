﻿#include <hip/hip_runtime.h>
#include <math.h>


// Kernel function to generate Mandelbrot set with colors
__global__ void mandelbrotWithColor(int *image, int width, int height, int max_iter,
                                    double center_real, double center_imag, double zoom) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    double aspect_ratio = (double)width / height;

    double real = ((x - width / 2.0) / (width * zoom)) * aspect_ratio + center_real;
    double imag = ((y - height / 2.0) / (height * zoom)) + center_imag;

    double zx = 0.0, zy = 0.0;
    int iteration = 0;
    while (zx * zx + zy * zy <= 4.0 && iteration < max_iter) {
        double temp = zx * zx - zy * zy + real;
        zy = 2.0 * zx * zy + imag;
        zx = temp;
        iteration++;
    }

    float hue = (float)(iteration % 256) / 255.0f;
    float saturation = 1.0f;
    float brightness = (iteration < max_iter) ? 1.0f : 0.0f;

    int i = (int)(hue * 6.0f);
    float f = (hue * 6.0f) - i;
    float p = brightness * (1.0f - saturation);
    float q = brightness * (1.0f - f * saturation);
    float t = brightness * (1.0f - (1.0f - f) * saturation);

    float r, g, b;
    switch (i % 6) {
        case 0: r = brightness; g = t; b = p; break;
        case 1: r = q; g = brightness; b = p; break;
        case 2: r = p; g = brightness; b = t; break;
        case 3: r = p; g = q; b = brightness; break;
        case 4: r = t; g = p; b = brightness; break;
        case 5: r = brightness; g = p; b = q; break;
    }

    int color = ((int)(r * 255) << 16) | ((int)(g * 255) << 8) | (int)(b * 255);
    image[y * width + x] = color;
}

// Exported functions to be used in C#
extern "C" {
    __declspec(dllexport) void runMandelbrotWithColor(
        int *image, int width, int height, int max_iter,
        double center_real, double center_imag, double zoom) {

        int *d_image;
        size_t size = width * height * sizeof(int);

        // Allocate memory on the device
        hipError_t err = hipMalloc(&d_image, size);
        if (err != hipSuccess) {
            // printf("CUDA malloc failed: %s\n", cudaGetErrorString(err));
            return;
        }

        // Launch the kernel
        dim3 blockDim(32, 8);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
                     (height + blockDim.y - 1) / blockDim.y);

        mandelbrotWithColor<<<gridDim, blockDim>>>(d_image, width, height, max_iter,
                                                   center_real, center_imag, zoom);

        // Check for launch errors
        err = hipGetLastError();
        if (err != hipSuccess) {
            // printf("Kernel launch failed: %s\n", cudaGetErrorString(err));
            hipFree(d_image);
            return;
        }

        // Copy the result back to the host
        err = hipMemcpy(image, d_image, size, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            // printf("CUDA memcpy failed: %s\n", cudaGetErrorString(err));
        }

        // Free the device memory
        hipFree(d_image);
    }
}
